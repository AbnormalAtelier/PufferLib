
#include <hip/hip_runtime.h>
__global__ void advantage_kernel(
    float* reward_block,    // [num_steps, horizon]
    float* reward_mask,     // [num_steps, horizon]
    float* values_mean,     // [num_steps, horizon]
    float* values_std,      // [num_steps, horizon]
    float* buf,            // [num_steps, horizon]
    float* dones,          // [num_steps]
    float* rewards,        // [num_steps]
    float* advantages,     // [num_steps]
    int* bounds,          // [num_steps]
    int num_steps,
    float r_std,
    int horizon
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_steps) return;

    int k = 0;
    for (int j = 0; j < horizon-1; j++) {
        int t = i + j;
        if (t >= num_steps - 1) {
            break;
        }
        if (dones[t+1]) {
            k++;
            break;
        }
        k++;
    }

    float gamma_max = 0.0f;
    float n = 0.0f;
    for (int j = k-1; j >= 0; j--) {
        int idx = i * horizon + j;
        n++;

        float vstd = values_std[idx];
        if (vstd == 0.0f) {
            buf[idx] = 0.0f;
            continue;
        }

        float gamma = 1.0f / (vstd*vstd);
        /*
        if (r_std != 0.0f) {
            gamma -= 1.0f/(r_std*r_std);
        }
        */

        if (gamma < 0.0f) {
            gamma = 0.0f;
        }

        if (gamma > gamma_max) {
            gamma_max = gamma;
        }
        buf[idx] = gamma;
        reward_mask[idx] = 1.0f;
    }

    float bootstrap = 0.0f;
    //if (k == horizon-1) {
    //    bootstrap = buf[i*horizon + horizon - 1]*values_mean[i*horizon + horizon - 1];
    //}

    float R = 0.0f;
    for (int j = k-1; j >= 0; j--) {
        int t = i + j;
        int idx = i * horizon + j;
        float r = rewards[t+1];

        float gamma = buf[idx];
        if (gamma_max > 0) {
            gamma /= gamma_max;
        }

        R += r*gamma;
        reward_block[idx] = r;
        buf[idx] = gamma;
    }

    advantages[i] = R - values_mean[i*horizon];
    bounds[i] = k;
}
