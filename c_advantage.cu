#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void advantage_kernel(
    float* reward_block,    // [num_steps, horizon]
    float* reward_mask,     // [num_steps, horizon]
    float* values_mean,     // [num_steps, horizon]
    float* values_std,      // [num_steps, horizon]
    float* buf,            // [num_steps, horizon]
    float* dones,          // [num_steps]
    float* rewards,        // [num_steps]
    float* advantages,     // [num_steps]
    int* bounds,          // [num_steps]
    int num_steps,
    float r_std,
    int horizon
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_steps) return;

    int k = 0;

    for (int j = 0; j < horizon; j++) {
        int t = i + j;
        if (t >= num_steps - 1 || dones[t]) {
            break;
        }
        k = j + 1;
    }

    float R = 0.0f;
    for (int j = k-2; j > 0; j--) {
        int t = i + j;
        int idx = i * horizon + j;

        float r = rewards[t+1];
        float vstd = values_std[idx];
        if (vstd == 0.0f) {
            buf[idx] = 0.0f;
            continue;
        }

        R += r/(vstd*vstd);
        if (r_std != 0.0f) {
            R -= r/(r_std*r_std);
        }
        reward_block[idx] = R;
        reward_mask[idx] = 1.0f;
        buf[idx] = R;
    }

    advantages[i] = R - values_mean[i*horizon];
    bounds[i] = k;
}

/*
__global__ void advantage_kernel(
    float* reward_block,    // [num_steps, horizon]
    float* reward_mask,     // [num_steps, horizon]
    float* values_mean,     // [num_steps, horizon]
    float* values_std,      // [num_steps, horizon]
    float* buf,            // [num_steps, horizon]
    float* dones,          // [num_steps]
    float* rewards,        // [num_steps]
    float* advantages,     // [num_steps]
    int* bounds,          // [num_steps]
    int num_steps,
    float r_std,
    int horizon
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_steps) return;

    int k = 0;

    for (int j = 0; j < horizon; j++) {
        int t = i + j;
        if (t >= num_steps - 1 || dones[t]) {
            break;
        }
        k = j + 1;
    }

    float R = 0.0f;
    for (int j = k-2; j > 0; j--) {
        int t = i + j;
        int idx = i * horizon + j;

        float r = rewards[t+1];
        float vstd = values_std[idx];
        if (vstd == 0.0f) {
            buf[idx] = 0.0f;
            continue;
        }

        R += r/(vstd*vstd);
        if (r_std != 0.0f) {
            R -= r/(r_std*r_std);
        }
        reward_block[idx] = R;
        reward_mask[idx] = 1.0f;
        buf[idx] = R;
    }

    advantages[i] = R - values_mean[i*horizon];
    bounds[i] = k;
}
*/

// Pybind11 module definition
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("advantage_kernel", [](torch::Tensor reward_block,
                                torch::Tensor reward_mask,
                                torch::Tensor values_mean,
                                torch::Tensor values_std,
                                torch::Tensor buf,
                                torch::Tensor dones,
                                torch::Tensor rewards,
                                torch::Tensor advantages,
                                torch::Tensor bounds,
                                int num_steps,
                                float vstd_max,
                                int horizon) {
        // Launch the kernel
        int threads_per_block = 256;
        int blocks = (num_steps + threads_per_block - 1) / threads_per_block;

        advantage_kernel<<<blocks, threads_per_block>>>(
            reward_block.data_ptr<float>(),
            reward_mask.data_ptr<float>(),
            values_mean.data_ptr<float>(),
            values_std.data_ptr<float>(),
            buf.data_ptr<float>(),
            dones.data_ptr<float>(),
            rewards.data_ptr<float>(),
            advantages.data_ptr<float>(),
            bounds.data_ptr<int>(),
            num_steps,
            vstd_max, 
            horizon
        );

        // Check for CUDA errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error(hipGetErrorString(err));
        }
    }, "Compute advantages with CUDA");
}
