#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace pufferlib {

/*
__global__ void p3o_kernel(
    float* reward_block,    // [num_steps, horizon]
    float* reward_mask,     // [num_steps, horizon]
    float* values_mean,     // [num_steps, horizon]
    float* values_std,      // [num_steps, horizon]
    float* buf,            // [num_steps, horizon]
    float* dones,          // [num_steps]
    float* rewards,        // [num_steps]
    float* advantages,     // [num_steps]
    int* bounds,          // [num_steps]
    int num_steps,
    float r_std,
    float puf,
    int horizon
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_steps) return;

    int k = 0;
    for (int j = 0; j < horizon-1; j++) {
        int t = i + j;
        if (t >= num_steps - 1) {
            break;
        }
        if (dones[t+1]) {
            k++;
            break;
        }
        k++;
    }

    float gamma_max = 0.0f;
    float n = 0.0f;
    for (int j = k-1; j >= 0; j--) {
        int idx = i * horizon + j;
        n++;

        float vstd = values_std[idx];
        if (vstd == 0.0f) {
            buf[idx] = 0.0f;
            continue;
        }

        float gamma = 1.0f / (vstd*vstd);
        if (r_std != 0.0f) {
            gamma -= puf/(r_std*r_std);
        }

        if (gamma < 0.0f) {
            gamma = 0.0f;
        }

        if (gamma > gamma_max) {
            gamma_max = gamma;
        }
        buf[idx] = gamma;
        reward_mask[idx] = 1.0f;
    }

    //float bootstrap = 0.0f;
    //if (k == horizon-1) {
    //    bootstrap = buf[i*horizon + horizon - 1]*values_mean[i*horizon + horizon - 1];
    //}

    float R = 0.0f;
    for (int j = 0; j <= k-1; j++) {
        int t = i + j;
        int idx = i * horizon + j;
        float r = rewards[t+1];

        float gamma = buf[idx];
        if (gamma_max > 0) {
            gamma /= gamma_max;
        }

        if (j >= 16 && values_std[idx] > 0.95*r_std) {
            break;
        }

        R += gamma * (r - values_mean[idx]);
        reward_block[idx] = r;
        buf[idx] = gamma;
    }

    advantages[i] = R;
    bounds[i] = k;
}


void compute_p3o(torch::Tensor reward_block, torch::Tensor reward_mask,
        torch::Tensor values_mean, torch::Tensor values_std, torch::Tensor buf,
        torch::Tensor dones, torch::Tensor rewards, torch::Tensor advantages,
        torch::Tensor bounds, int num_steps, float vstd_max, float puf,
        int horizon) {

    // TODO: Port from python
    assert all(t.is_cuda for t in [reward_block, reward_mask, values_mean, values_std, 
                                  buf, dones, rewards, advantages, bounds]), "All tensors must be on GPU"
    
    # Ensure contiguous memory
    tensors = [reward_block, reward_mask, values_mean, values_std, buf, dones, rewards, advantages, bounds]
    for t in tensors:
        t.contiguous()
        assert t.is_cuda

    num_steps = rewards.shape[0]
    
    # Precompute vstd_min and vstd_max
    #vstd_max = values_std.max().item()
    #vstd_min = values_std.min().item()

    # Launch kernel
    threads_per_block = 256
    assert num_steps % threads_per_block == 0
    blocks = (num_steps + threads_per_block - 1) // threads_per_block
 
    // Launch the kernel
    int threads_per_block = 256;
    int blocks = (num_steps + threads_per_block - 1) / threads_per_block;

    p3o_kernel<<<blocks, threads_per_block>>>(
        reward_block.data_ptr<float>(),
        reward_mask.data_ptr<float>(),
        values_mean.data_ptr<float>(),
        values_std.data_ptr<float>(),
        buf.data_ptr<float>(),
        dones.data_ptr<float>(),
        rewards.data_ptr<float>(),
        advantages.data_ptr<float>(),
        bounds.data_ptr<int>(),
        num_steps,
        vstd_max, 
        puf,
        horizon
    );

    // Check for CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
    return;
}

// [num_steps, horizon]
__global__ void gae_kernel(float* values, float* rewards, float* dones,
        float* advantages, float gamma, float gae_lambda, int num_steps, int horizon) {
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int offset = row*horizon;
    gae_row(values + offset, rewards + offset, dones + offset,
        advantages + offset, gamma, gae_lambda, horizon);
}

torch::Tensor compute_gae(torch::Tensor values, torch::Tensor rewards,
        torch::Tensor dones, float gamma, float gae_lambda) {
    int num_steps = values.size(0);
    int horizon = values.size(1);
    torch::Tensor advantages = gae_check(values, rewards, dones, num_steps, horizon);
    TORCH_CHECK(values.is_cuda(), "All tensors must be on GPU");

    int threads_per_block = 256;
    int blocks = (num_steps + threads_per_block - 1) / threads_per_block;
    assert(num_steps % threads_per_block == 0);

    gae_kernel<<<blocks, threads_per_block>>>(
        values.data_ptr<float>(),
        rewards.data_ptr<float>(),
        dones.data_ptr<float>(),
        advantages.data_ptr<float>(),
        gamma,
        gae_lambda,
        num_steps,
        horizon
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return advantages;
}

 // [num_steps, horizon]
__global__ void vtrace_kernel(float* values, float* rewards, float* dones, float* importance,
        float* vs, float* advantages, float gamma, float rho_clip, float c_clip, int num_steps, int horizon) {
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int offset = row*horizon;
    vtrace_row(values + offset, rewards + offset, dones + offset,
        importance + offset, vs + offset, advantages + offset, gamma, rho_clip, c_clip, horizon);
}

void compute_vtrace(torch::Tensor values, torch::Tensor rewards,
        torch::Tensor dones, torch::Tensor importance, torch::Tensor vs, torch::Tensor advantages,
        float gamma, float rho_clip, float c_clip) {
    int num_steps = values.size(0);
    int horizon = values.size(1);
    vtrace_check(values, rewards, dones, importance, vs, advantages, num_steps, horizon);
    TORCH_CHECK(values.is_cuda(), "All tensors must be on GPU");
    assert(horizon <= max_horizon);

    int threads_per_block = 128;
    int blocks = (num_steps + threads_per_block - 1) / threads_per_block;
    assert(num_steps % threads_per_block == 0);

    vtrace_kernel<<<blocks, threads_per_block>>>(
        values.data_ptr<float>(),
        rewards.data_ptr<float>(),
        dones.data_ptr<float>(),
        importance.data_ptr<float>(),
        vs.data_ptr<float>(),
        advantages.data_ptr<float>(),
        gamma,
        rho_clip,
        c_clip,
        num_steps,
        horizon
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
}
*/

static const int max_horizon = 256;
__host__ __device__ void puff_advantage_row_cuda(float* values, float* rewards, float* dones,
        float* importance, float* vs, float* advantages, float gamma, float lambda,
        float rho_clip, float c_clip, int horizon) {
    vs[horizon-1] = values[horizon-1];
    float lastpufferlam = 0;
    for (int t = horizon-2; t >= 0; t--) {
        int t_next = t + 1;
        float nextnonterminal = 1.0 - dones[t_next];
        float rho_t = fminf(importance[t], rho_clip);
        float c_t = fminf(importance[t], c_clip);
        // TODO: t_next works and t doesn't. Check original formula
        float delta = rho_t*(rewards[t_next] + gamma*values[t_next]*nextnonterminal - values[t]);
        lastpufferlam = delta + gamma*lambda*c_t*lastpufferlam*nextnonterminal;
        
        //float delta = rewards[t_next] + gamma*values[t_next]*nextnonterminal - values[t];
        //lastpufferlam = delta + gamma*lambda*lastpufferlam*nextnonterminal;


        advantages[t] = lastpufferlam;
        vs[t] = advantages[t] + values[t];
        //advantages[t] = rho_t*(rewards[t] + gamma*vs[t_next]*nextnonterminal - values[t]);
        //vs[t] = lastpufferlam + values[t];
    }
}

void vtrace_check_cuda(torch::Tensor values, torch::Tensor rewards,
        torch::Tensor dones, torch::Tensor importance, torch::Tensor vs, torch::Tensor advantages,
        int num_steps, int horizon) {

    // Validate input tensors
    torch::Device device = values.device();
    for (const torch::Tensor& t : {values, rewards, dones, importance, vs, advantages}) {
        TORCH_CHECK(t.dim() == 2, "Tensor must be 2D");
        TORCH_CHECK(t.device() == device, "All tensors must be on same device");
        TORCH_CHECK(t.size(0) == num_steps, "First dimension must match num_steps");
        TORCH_CHECK(t.size(1) == horizon, "Second dimension must match horizon");
        TORCH_CHECK(t.dtype() == torch::kFloat32, "All tensors must be float32");
        assert(horizon <= max_horizon);
        if (!t.is_contiguous()) {
            t.contiguous();
        }
    }
}


 // [num_steps, horizon]
__global__ void puff_advantage_kernel(float* values, float* rewards, float* dones, float* importance,
        float* vs, float* advantages, float gamma, float lambda,
        float rho_clip, float c_clip, int num_steps, int horizon) {
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int offset = row*horizon;
    puff_advantage_row_cuda(values + offset, rewards + offset, dones + offset,
        importance + offset, vs + offset, advantages + offset, gamma, lambda, rho_clip, c_clip, horizon);
}

void compute_puff_advantage_cuda(torch::Tensor values, torch::Tensor rewards,
        torch::Tensor dones, torch::Tensor importance, torch::Tensor vs, torch::Tensor advantages,
        double gamma, double lambda, double rho_clip, double c_clip) {
    int num_steps = values.size(0);
    int horizon = values.size(1);
    vtrace_check_cuda(values, rewards, dones, importance, vs, advantages, num_steps, horizon);
    TORCH_CHECK(values.is_cuda(), "All tensors must be on GPU");
    assert(horizon <= max_horizon);

    int threads_per_block = 256;
    if (threads_per_block > num_steps) {
        threads_per_block = 2*(num_steps/2);
    }
    int blocks = (num_steps + threads_per_block - 1) / threads_per_block;
    assert(num_steps % threads_per_block == 0);

    puff_advantage_kernel<<<blocks, threads_per_block>>>(
        values.data_ptr<float>(),
        rewards.data_ptr<float>(),
        dones.data_ptr<float>(),
        importance.data_ptr<float>(),
        vs.data_ptr<float>(),
        advantages.data_ptr<float>(),
        gamma,
        lambda,
        rho_clip,
        c_clip,
        num_steps,
        horizon
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
}

TORCH_LIBRARY_IMPL(pufferlib, CUDA, m) {
  m.impl("compute_puff_advantage", &compute_puff_advantage_cuda);
}

}
